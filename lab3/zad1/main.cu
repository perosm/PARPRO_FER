
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <chrono>

#define N (1 << 20) // vrijednost potencije
#define L 256 // veličina grupe dretvi -----> threads per block


//#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
//#else
//__device__ double atomicAdd(double* address, double val) {
//    unsigned long long int* address_as_ull = (unsigned long long int*)address;
//    unsigned long long int old = *address_as_ull, assumed;
//
//    do {
//        assumed = old;
//        old = atomicCAS(address_as_ull, assumed,
//            __double_as_longlong(val + __longlong_as_double(assumed)));
//    } while (assumed != old);
//
//    return __longlong_as_double(old);
//}
//#endif



using namespace std;

__global__ void count_primes(int *device_array, int* device_prime_count, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    for (int i = idx; i < n; i += stride) {
        bool is_prime = true;
        int num = device_array[i];
        if (num > 1){
            for (int j = 2; j < num; j++){
                if (num % j == 0) {
                    is_prime = false;
                    break;
                }
            }
            if(is_prime){
            	*device_prime_count += 1;
                //atomicAdd(device_prime_count, 1);
            }
        }
    }
}

int main() { // nvcc main.cu -o main_cu
    int *host_array = (int*) malloc(N * sizeof(int)); // brojevi

    for (int i = 0; i < N; i++) {
        host_array[i] = i + 1;
    }
    chrono::duration<double> min_duration = std::chrono::duration_cast<std::chrono::seconds>(std::chrono::seconds::max() - std::chrono::seconds::min());
    
    int threadsPerBlock_min = -1;
    int blocksPerGrid_min = -1;
	int *device_array;
	hipMalloc(&device_array, N * sizeof(int));
	hipMemcpy(device_array, host_array, N * sizeof(int), hipMemcpyHostToDevice);
	int *device_prime_counter;
	for(int l = 1; l <= L; l*=2){
		int host_prime_count = 0;

		// zauzimanje memorije na device-u
		hipMalloc(&device_prime_counter, sizeof(int));
		// kopiranje vektora iz host memorije u device memoriju
		hipMemcpy(device_prime_counter, &host_prime_count, sizeof(int), hipMemcpyHostToDevice);

		// pozivanje kernela
		int threadsPerBlock = l;
		int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
		auto start = chrono::high_resolution_clock::now();
		
		count_primes<<<blocksPerGrid, threadsPerBlock>>>(device_array, device_prime_counter, N);
		
		// sinkronizacija
		//cudaDeviceSynchronize();

		auto end = chrono::high_resolution_clock::now();
		chrono::duration<double> duration = end - start;

		if(duration < min_duration || min_duration.count() == -1){
			min_duration = duration;
			blocksPerGrid_min = blocksPerGrid;
			threadsPerBlock_min = threadsPerBlock;
		}
		// kopiranje rezultata iz device u host memoriju
		hipMemcpy(&host_prime_count, device_prime_counter, sizeof(int), hipMemcpyDeviceToHost);
		
		cout << "Broj blokova: " << blocksPerGrid << " , broj dretvi po bloku: " << threadsPerBlock << endl;
		cout << "Trajanje (sek): " << duration.count() << endl;
		cout << "Broj prim brojeva: " << host_prime_count << endl;

		hipFree(device_array);
		hipFree(device_prime_counter);
	}
	free(host_array);
	
	cout << endl;
	cout << "Broj blokova: " << blocksPerGrid_min << " , broj dretvi po bloku: " << threadsPerBlock_min << endl;
	cout << "Trajanje (sek): " << min_duration.count() << endl;

    return 0;
}
